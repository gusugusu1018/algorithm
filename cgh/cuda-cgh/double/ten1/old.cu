#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <cstdint>
#include <sys/time.h>
#include <sys/resource.h>
#include <hip/hip_runtime.h>
#include "bitmap.hpp"

__global__ void distance_gpu(int *x_d,int *y_d,float *z_d,float *img_buf_d,int *points_d,int _width)
{
   int i,j,k;

   i=blockIdx.x;
   j=threadIdx.x;

   float kankaku,hatyou,goukei,pi;

   hatyou=0.633F;
   kankaku=10.5F;
   pi=3.14159265F;
   goukei=2.0F*pi*kankaku/hatyou;

   float dx,dy,tmp;

   tmp=0.0F;

   for(k=0;k<*points_d;k++){
      dx=(float)(x_d[k]-j);
      dy=(float)(y_d[k]-i);
      tmp=tmp+cos(goukei*0.5F*(dx*dx+dy*dy)/z_d[k]);
   }
   img_buf_d[i*_width+j] = tmp;
}

int main(){
   const static int width = 1024;
   const static int height = 1024;
   int points;
   FILE *fp;
   int i,j;
   Bitmap bmp(width, height);

   fp=fopen("../../../data/ten1.3d","rb");
   if(fp==NULL){
      printf("Can't open file\n");
   }

   fread(&points,sizeof(int),1,fp);

   int x[points];
   int y[points];
   float z[points];
   int *points_d;
   hipMemcpy(points_d,&points,sizeof(int),hipMemcpyHostToDevice);

   int *x_d,*y_d;
   float *z_d;
   float *img_buf_d;

   dim3 blocks(1024,1,1);
   dim3 threads(1024,1,1);

   int x_buf,y_buf,z_buf;

   for(i=0;i<points;i++){
      fread(&x_buf,sizeof(int),1,fp);
      fread(&y_buf,sizeof(int),1,fp);
      fread(&z_buf,sizeof(int),1,fp);

      x[i]=x_buf*40+960;
      y[i]=y_buf*40+540;
      z[i]=((float)z_buf)*40+100000.0;
   }
   fclose(fp);

   hipMalloc((void**)&x_d,points*sizeof(int));
   hipMalloc((void**)&y_d,points*sizeof(int));
   hipMalloc((void**)&z_d,points*sizeof(float));

   hipMalloc((void**)&img_buf_d,width*height*sizeof(float));

   float *img_buf;

   img_buf=(float *)malloc(sizeof(float)*width*height);
   for(i=0;i<width*height;i++){
      img_buf[i]=0.0f;
   }

   hipMemcpy(x_d,x,points*sizeof(int),hipMemcpyHostToDevice);
   hipMemcpy(y_d,y,points*sizeof(int),hipMemcpyHostToDevice);
   hipMemcpy(z_d,z,points*sizeof(float),hipMemcpyHostToDevice);

   hipMemcpy(img_buf_d,img_buf,width*height*sizeof(float),hipMemcpyHostToDevice);

   distance_gpu<<<blocks,threads>>>(x_d,y_d,z_d,img_buf_d,points_d,width);

   hipMemcpy(img_buf,img_buf_d,width*height*sizeof(float),hipMemcpyDeviceToHost);

   float min,max,mid;

   min=img_buf[0];
   max=img_buf[0];

   for(i=0;i<height;i++){
      for(j=0;j<width;j++){
         if(min>img_buf[i*width+j]){
            min=img_buf[i*width+j];
         }
         if(max<img_buf[i*width+j]){
            max=img_buf[i*width+j];
         }
      }
   }

   mid=0.5*(min+max);

   printf("min = %lf  max = %lf  mid = %lf\n",min,max,mid);


   unsigned char *img;
   img=(unsigned char *)malloc(sizeof(unsigned char)*width*height);

   for(i=0;i<width*height;i++){
      if(img_buf[i]<mid){
         img[i]=0;
      }
      if(img_buf[i]>mid){
         img[i]=255;
      }
   }

   FILE *bmpFp;
   bmpFp=fopen("cgh.bmp","wb");
   if(bmpFp==NULL){
      printf("Can't open file\n");
   }
   bmp.write(img,bmpFp);

   free(img);
   free(img_buf);
   fclose(bmpFp);
   hipFree(points_d);
   hipFree(x_d);
   hipFree(y_d);
   hipFree(z_d);
   hipFree(img_buf_d);

   return 0;
}
