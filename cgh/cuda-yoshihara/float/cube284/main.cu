#include <stdio.h>
#include <math.h>
#include<stdint.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define WIDTH 1024
#define HEIGHT 1024
#define POINTS 284

#pragma pack(push,1)
typedef struct tagBITMAPFILEHEADER
{
   unsigned short bfType;
   uint32_t  bfSize;
   unsigned short bfReserved1;
   unsigned short bfReserved2;
   uint32_t  bf0ffBits;
}BITMAPFILEHEADER;
#pragma pack(pop)

typedef struct tagBITMAPINFOHEADER
{
   uint32_t  biSize;
   int32_t	biWidth;
   int32_t	biHeight;
   unsigned short  biPlanes;
   unsigned short  biBitCount;
   uint32_t   biCompression;
   uint32_t   biSizeImage;
   int32_t	 biXPelsPerMeter;
   int32_t	 biYPelsPerMeter;
   uint32_t   biCirUsed;
   uint32_t   biCirImportant;
}BITMAPINFOHEADER;

typedef struct tagRGBQUAD
{
   unsigned char  rgbBlue;
   unsigned char  rgbGreen;
   unsigned char  rgbRed;
   unsigned char  rgbReserved;
}RGBQUAD;

typedef struct tagBITMAPINFO
{
   BITMAPINFOHEADER bmiHeader;
   RGBQUAD          bmiColors[1];
}BITMAPINFO;


__global__ void distance_gpu(int *x_d,int *y_d,float *z_d,float *img_buf_d)
{
   int i = blockDim.x*blockIdx.x+threadIdx.x;
   int j = blockDim.y*blockIdx.y+threadIdx.y;
   int k;
   float coef=49.63021f;
   for(k=0;k<284;k++){
   //for(k=0;k<POINTS;k++){
      img_buf_d[i*WIDTH+j]=img_buf_d[i*WIDTH+j]+cos(coef*float((j-x_d[k])*(j-x_d[k])+(i-y_d[k])*(i-y_d[k]))/z_d[k]);
   }
}

int main(){
   BITMAPFILEHEADER    BmpFileHeader;
   BITMAPINFOHEADER    BmpInfoHeader;
   RGBQUAD             RGBQuad[256];

   FILE *datafp;
   int i,j;

   BmpFileHeader.bfType                =19778;
   BmpFileHeader.bfSize                =14+40+1024+(WIDTH*HEIGHT);
   BmpFileHeader.bfReserved1           =0;
   BmpFileHeader.bfReserved2           =0;
   BmpFileHeader.bf0ffBits             =14+40+1024;

   BmpInfoHeader.biSize                =40;
   BmpInfoHeader.biWidth               =WIDTH;
   BmpInfoHeader.biHeight              =HEIGHT;
   BmpInfoHeader.biPlanes              =1;
   BmpInfoHeader.biBitCount            =8;     //256階調
   BmpInfoHeader.biCompression         =0L;
   BmpInfoHeader.biSizeImage           =0L;
   BmpInfoHeader.biXPelsPerMeter       =0L;
   BmpInfoHeader.biYPelsPerMeter       =0L;
   BmpInfoHeader.biCirUsed             =0L;
   BmpInfoHeader.biCirImportant        =0L;

   for(i=0;i<256;i++){
      RGBQuad[i].rgbBlue                =i;
      RGBQuad[i].rgbGreen               =i;
      RGBQuad[i].rgbRed                 =i;
      RGBQuad[i].rgbReserved            =0;
   }

   datafp=fopen("../../../data/cube284.3d","rb");
   if(datafp==NULL){
      printf("Can't open data file\n");
      return 1;
   }
   int x[POINTS];
   int y[POINTS];
   float z[POINTS];
   int x_buf,y_buf,z_buf;
   for(i=0;i<POINTS;i++){
      fread(&x_buf,sizeof(int),1,datafp);
      fread(&y_buf,sizeof(int),1,datafp);
      fread(&z_buf,sizeof(int),1,datafp);
      x[i]=x_buf*40+512;
      y[i]=y_buf*40+512;
      z[i]=((float)z_buf)*40+100000.0;
   }
   fclose(datafp);
   int *x_d,*y_d;
   float *z_d;
   float *img_buf_d;
   hipMalloc((void**)&x_d,POINTS*sizeof(int));
   hipMalloc((void**)&y_d,POINTS*sizeof(int));
   hipMalloc((void**)&z_d,POINTS*sizeof(float));
   hipMalloc((void**)&img_buf_d,WIDTH*HEIGHT*sizeof(float));
   float *img_buf;
   img_buf=(float *)malloc(sizeof(float)*WIDTH*HEIGHT);
   for(i=0;i<WIDTH*HEIGHT;i++){
      img_buf[i]=0.0f;
   }
   hipMemcpy(x_d,x,POINTS*sizeof(int),hipMemcpyHostToDevice);
   hipMemcpy(y_d,y,POINTS*sizeof(int),hipMemcpyHostToDevice);
   hipMemcpy(z_d,z,POINTS*sizeof(float),hipMemcpyHostToDevice);
   hipMemcpy(img_buf_d,img_buf,WIDTH*HEIGHT*sizeof(float),hipMemcpyHostToDevice);

   dim3 threads(1,1,1);
   dim3 blocks(ceil(WIDTH/threads.x),ceil(HEIGHT/threads.y),1);
   distance_gpu<<<blocks,threads>>>(x_d,y_d,z_d,img_buf_d);
   hipMemcpy(img_buf,img_buf_d,WIDTH*HEIGHT*sizeof(float),hipMemcpyDeviceToHost);
   float min,max,mid;
   min=img_buf[0];
   max=img_buf[0];
   for(i=0;i<HEIGHT;i++){
      for(j=0;j<WIDTH;j++){
         if(min>img_buf[i*WIDTH+j]){
            min=img_buf[i*WIDTH+j];
         }
         if(max<img_buf[i*WIDTH+j]){
            max=img_buf[i*WIDTH+j];
         }
      }
   }
   mid=0.5f*(min+max);
   printf("min = %lf  max = %lf  mid = %lf\n",min,max,mid);

   if (min != -70.310204f) {
      printf("min caliculation failure\n");
      return 1;
   }
   if (max != 75.139923f) {
      printf("max caliculation failure\n");
      return 1;
   }

   unsigned char *img;
   img=(unsigned char *)malloc(sizeof(unsigned char)*WIDTH*HEIGHT);
   for(i=0;i<WIDTH*HEIGHT;i++){
      if(img_buf[i]<mid){
         img[i]=0;
      }
      if(img_buf[i]>mid){
         img[i]=255;
      }
   }
   FILE *bmpfp;
   bmpfp=fopen("cgh.bmp","wb");
   if(bmpfp==NULL){
      printf("Can't write file\n");
      return 1;
   }

   fwrite(&BmpFileHeader, sizeof(BmpFileHeader) , 1 ,bmpfp);
   fwrite(&BmpInfoHeader, sizeof(BmpInfoHeader) , 1 ,bmpfp);
   fwrite(&RGBQuad[0], sizeof(RGBQuad[0]) , 256 ,bmpfp);
   fwrite(img,sizeof(unsigned char),WIDTH*HEIGHT,bmpfp);
   fclose(bmpfp);

   free(img);
   free(img_buf);

   hipFree(x_d);
   hipFree(y_d);
   hipFree(z_d);
   hipFree(img_buf_d);

   return 0;
}
