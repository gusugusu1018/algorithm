#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define N_X 100
#define N_Y 100
#define N_EPOCH 3
#define POINTS 1600

__global__ void calc_potential(const int *_ob_x,const int *_ob_y,const int p_x,const int p_y,const int g_x,const int g_y,float *U_d) {
   int i = blockDim.x*blockIdx.x+threadIdx.x;
   int j = blockDim.y*blockIdx.y+threadIdx.y;
   int k,m;
   for(k=0;k<=1000;k++) {
      /*
      for (m=0;m<POINTS;m++) {
         if (_ob_x[m]==i&&_ob_y[m]==j) U_d[i*N_X+j]=100.0f;
      }
      */
      if ((p_x==i&&p_y==j)|| i==0 || j==0 || i==(N_X-1) || j==(N_Y-1)) {
         U_d[i*N_X+j]=100.0f;
      } else if (g_x==i&&g_y==j) {
         U_d[i*N_X+j]=-100.0f;
      } else {
         U_d[i*N_X+j]=0.25f*(U_d[(i+1)*N_X+j]+U_d[(i-1)*N_X+j]+U_d[i*N_X+j+1]+U_d[i*N_X+j-1]);
      }
      __syncthreads();
   }
}

float U[N_X*N_Y]={0.0f};

double gettimeofday_sec(){
   struct timeval tv;
   gettimeofday(&tv, NULL);
   return tv.tv_sec + tv.tv_usec * 1e-6;
}

void get_obstacle(int x[POINTS],int y[POINTS]);
void destination(int p_x,int p_y,int *min_x,int *min_y);

int main()
{
   int i,j,n,m;
   n=0;
   int ob_x[POINTS];
   int ob_y[POINTS];
   int p_x = 20;
   int p_y = 20;
   int g_x = 40;
   int g_y = 80;
   int dst_x;
   int dst_y;
   get_obstacle(ob_x,ob_y);
   FILE *potential;
   potential=fopen("potential.data","w");
   FILE *path;
   path=fopen("path.data","w");
   double start,end;

   int *_ob_x,*_ob_y;
   float *U_d;
   hipMalloc((void**)&_ob_x,POINTS*sizeof(int));
   hipMalloc((void**)&_ob_y,POINTS*sizeof(int));
   hipMalloc((void**)&U_d,N_X*N_Y*sizeof(float));
   dim3 threads(32,32,1);
   dim3 blocks(ceil(N_X/threads.x),ceil(N_Y/threads.y),1);
   hipMemcpy(_ob_x, ob_x, POINTS*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(_ob_y, ob_y, POINTS*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(U_d,U,N_X*N_Y*sizeof(float), hipMemcpyHostToDevice);

   //   for (n=0;n<N_EPOCH;n++) {
   hipDeviceSynchronize();
   start=gettimeofday_sec();

   calc_potential<<<blocks, threads>>>(_ob_x,_ob_y,p_x,p_y,g_x,g_y,U_d);

   hipDeviceSynchronize();
   end=gettimeofday_sec();
   hipMemcpy(U,U_d,N_X*N_Y*sizeof(float), hipMemcpyDeviceToHost);
   hipFree(_ob_x);
   hipFree(_ob_y);
   hipFree(U_d);
   hipDeviceReset();
   printf("time\t%lf s\n", end-start);
   /*
   destination(p_x,p_y,&dst_x,&dst_y);
   if (U[dst_x*N_X+dst_y]>100.0f) {
      return 1;
   }
   fprintf(path,"%d %d %d\n",dst_x,dst_y,100-n);
   if ((dst_x==p_x)&&(dst_y==p_y)) break;
   p_x=dst_x;
   p_y=dst_y;
   */
   //   }
   /*
   for (m=0;m<POINTS;m++) U[ob_x[m]*N_X+ob_y[m]]=100.0f;
   U[p_x*N_X+p_y]=100.0f;
   U[g_x*N_X+g_y]=-100.0f;
   */
   for(i=0;i<N_X;i++) {
      for(j=0;j<N_Y;j++) {
         fprintf(potential,"%d %d %f\n",i,j,U[i*N_X+j]);
      }
      fprintf(potential,"\n");
   }
   fclose(potential);
   fclose(path);
   return 0;
}

void get_obstacle(int x[POINTS],int y[POINTS]) {
   int i;
   FILE *readfp;
   if ((readfp = fopen("POINTS.2d", "rb")) == NULL) {
      printf("Can't open a file\n");
      exit(1);
   }
   int x_buf;
   int y_buf;
   for (i=0;i<POINTS;i++) {
      fread(&x_buf,sizeof(int),1,readfp);
      fread(&y_buf,sizeof(int),1,readfp);
      x[i]=x_buf;
      y[i]=y_buf;
   }
   fclose(readfp);
}

void destination(int p_x,int p_y,int *min_x,int *min_y) {
   *min_x=p_x;
   *min_y=p_y;
   if (U[p_x*N_X+(p_y-1)]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x;
      *min_y=p_y-1;
   }
   if (U[p_x*N_X+(p_y+1)]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x;
      *min_y=p_y+1;
   }
   if (U[(p_x-1)*N_X+p_y]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x-1;
      *min_y=p_y;
   }
   if (U[(p_x+1)*N_X+p_y]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x+1;
      *min_y=p_y;
   }
   if (U[(p_x+1)*N_X+(p_y+1)]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x+1;
      *min_y=p_y+1;
   }
   if (U[(p_x+1)*N_X+(p_y-1)]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x+1;
      *min_y=p_y-1;
   }
   if (U[(p_x-1)*N_X+(p_y+1)]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x-1;
      *min_y=p_y+1;
   }
   if (U[(p_x-1)*N_X+(p_y-1)]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x-1;
      *min_y=p_y-1;
   }
   if (U[p_x*N_X+(p_y-2)]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x;
      *min_y=p_y-2;
   }
   if (U[p_x*N_X+(p_y+2)]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x;
      *min_y=p_y+2;
   }
   if (U[(p_x-2)*N_X+p_y]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x-2;
      *min_y=p_y;
   }
   if (U[(p_x+2)*N_X+p_y]<U[*min_x*N_X+*min_y]) {
      *min_x=p_x+2;
      *min_y=p_y;
   }
}
